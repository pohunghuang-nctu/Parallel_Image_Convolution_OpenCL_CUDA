#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hostFE.h"
//#include "helper.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ float F3[9];
__constant__ float F5[25];
__constant__ float F7[49];

__global__ void convolution(const float* inputImage, float* outputImage, 
                            int imageHeight, int imageWidth, 
                            int filterWidth, int pitch_width_in, int pitch_width_out)
{
    const float *filter;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (filterWidth == 3) {
        filter = F3;
    } else if (filterWidth == 5) {
        filter = F5;
    } else if (filterWidth == 7) {
        filter = F7;
    } else  {}

    int halffilterSize = filterWidth / 2;
    float sum = 0;

    if(i < imageHeight && j < imageWidth) {
        for (int k = -halffilterSize; k <= halffilterSize; k++)
        {
            for (int l = -halffilterSize; l <= halffilterSize; l++)
            {
                if (i + k >= 0 && i + k < imageHeight &&
                    j + l >= 0 && j + l < imageWidth)
                {
                    sum += inputImage[(i + k) * pitch_width_in + j + l] *
                           filter[(k + halffilterSize) * filterWidth +
                                  l + halffilterSize];
                }
            }
        }
        outputImage[i * pitch_width_out + j] = sum;
    }
}

extern "C" void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage)
{


    int filterSize = filterWidth * filterWidth;

    if (filterWidth == 3)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(F3), filter, filterSize * sizeof(float));
    }
    else if (filterWidth == 5)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(F5), filter, filterSize * sizeof(float));
    }
    else if (filterWidth == 7)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(F7), filter, filterSize * sizeof(float));
    }
    else
    {
        printf("Filter width not supported\n");
        exit(1);
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    float *dev_input, *dev_filter, *dev_output;
    size_t pitch_in, pitch_out;
    int dev_width_in, dev_width_out;
    
    // input memory allocation
    hipMallocPitch((void**)&dev_input, &pitch_in, imageWidth * sizeof(float), imageHeight);
    dev_width_in = pitch_in / sizeof(float);
    printf("input pitch: %zu, width: %d\n", pitch_in, dev_width_in);
    fflush(stdout);
    // filter memory allocation
    hipMalloc((void**)&dev_filter, filterSize * sizeof(float));

    // output memory allocation
    hipMallocPitch((void**)&dev_output, &pitch_out, imageWidth * sizeof(float), imageHeight);
    dev_width_out = pitch_out / sizeof(float);
    printf("output pitch: %zu, width: %d\n", pitch_out, dev_width_out);
    fflush(stdout);    

    // Copy input array data to GPU
    hipMemcpy2D(dev_input, pitch_in, inputImage, imageWidth * sizeof(float), imageWidth * sizeof(float), imageHeight, hipMemcpyHostToDevice);
    hipMemcpy(dev_filter, filter, filterSize * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    dim3 dimBlock(16, 16); // block size, 25x25 threads per block
    dim3 dimGrid(ceil(imageWidth/16.0), ceil(imageHeight/16.0)); // grid size, depending on the image size and block size

    // Launch the Kernel on the GPU

    convolution<<<dimGrid, dimBlock>>>(dev_input, dev_output, imageHeight, imageWidth, filterWidth, dev_width_in, dev_width_out);

    // CUDA Device Synchronize 
    hipDeviceSynchronize();

    // Copy the results back to the host
    hipMemcpy2D(outputImage, imageWidth * sizeof(float), dev_output, pitch_out, imageWidth * sizeof(float), imageHeight, hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(dev_input);
    hipFree(dev_filter);
    hipFree(dev_output);
}
